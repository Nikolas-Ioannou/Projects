#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f
#define BLOCK_SIZE 256

typedef struct { float x, y, z, vx, vy, vz; } Body;

__global__ void bodyForce(Body *p, float dt, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;
    __shared__ Body sharedBodies[BLOCK_SIZE];
    for (int tile = 0; tile < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; tile++) {
        int index = tile * BLOCK_SIZE + threadIdx.x;
        if (index < n)
            sharedBodies[threadIdx.x] = p[index];
        __syncthreads();
        for (int j = 0; j < BLOCK_SIZE && (tile * BLOCK_SIZE + j) < n; j++) {
            float dx = sharedBodies[j].x - p[idx].x;
            float dy = sharedBodies[j].y - p[idx].y;
            float dz = sharedBodies[j].z - p[idx].z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;
            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }
        __syncthreads();
    }
    p[idx].vx += dt * Fx;
    p[idx].vy += dt * Fy;
    p[idx].vz += dt * Fz;
}

__global__ void integratePosition(Body *p, float dt, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    
    p[idx].x += p[idx].vx * dt;
    p[idx].y += p[idx].vy * dt;
    p[idx].z += p[idx].vz * dt;
}

int main(const int argc, const char** argv) {
    int nBodies = 2 << 11;
    if (argc > 1) nBodies = 2 << atoi(argv[1]);

    const char *initialized_values, *solution_values;
    if (nBodies == 2 << 11) {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } else {
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }
    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f;
    const int nIters = 10;
    int bytes = nBodies * sizeof(Body);
    float *buf = (float*)malloc(bytes);
    Body *p = (Body*)buf;
    read_values_from_file(initialized_values, buf, bytes);

    Body *d_p;
    hipMalloc((void**)&d_p, bytes);
    hipMemcpy(d_p, p, bytes, hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (nBodies + blockSize - 1) / blockSize;
    
    double totalTime = 0.0;
    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();
        bodyForce<<<gridSize, blockSize>>>(d_p, dt, nBodies);
        hipDeviceSynchronize();
        integratePosition<<<gridSize, blockSize>>>(d_p, dt, nBodies);
        hipDeviceSynchronize();
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }
    
    hipMemcpy(p, d_p, bytes, hipMemcpyDeviceToHost);
    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
    write_values_to_file(solution_values, buf, bytes);
    
    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);
    free(buf);
    hipFree(d_p);
}
